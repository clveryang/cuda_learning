#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <cstdio>

#define thread_per_block 256

__global__ void reduce_0(float * device_input, float * device_output){

    __shared__ float data[thread_per_block];
    float * block_pointer = device_input + 2 * blockIdx.x * blockDim.x;
    
    int thread_id = threadIdx.x;
    
    data[thread_id] = block_pointer[thread_id + blockDim.x] + block_pointer[thread_id];
    
    __syncthreads();

    for (int i = blockDim.x / 2; i > 32; i/=2)
    {
        
        if(thread_id < i){
            
            //int index = 2 * thread_id * i;
            data[thread_id] += data[thread_id + i];

        }
        __syncthreads();
    }

    if (thread_id < 32){
        data[thread_id] += data[thread_id + 32];
        data[thread_id] += data[thread_id + 16];
        data[thread_id] += data[thread_id + 8];
        data[thread_id] += data[thread_id + 4];
        data[thread_id] += data[thread_id + 2];
        data[thread_id] += data[thread_id + 1];
    }

    if (thread_id == 0)
        device_output[blockIdx.x] = data[0];


    /*
    减少blockIdx.x的一半

    */

}



bool check(float *gpu_result, float * cpu_result, size_t N){

    for(int i = 0; i < N; i++){

        if(abs(gpu_result[i] - cpu_result[i]) > 0.005)
            return false;

    }
    return true;
}

int main(){

    // 生成随机数
    size_t num_float = 48 * 1024 * 1024;

    float * host_input = (float*)malloc(num_float * sizeof(float));

    for(int i = 0; i < num_float; i++)
    {
        host_input[i] = 2.0 * (float)drand48() -1.0;
        //printf("host_input[%d]:%f \n", i, host_input[i]);
    }

    //降低block数量为原来的一半，block里面的线程编程不变，一个线程做线程搬运和一次加法

    int block = num_float / thread_per_block / 2;

    // 分配device空间
    float * device_input;
    hipMalloc((void**)&device_input, num_float*sizeof(float));
    hipMemcpy(device_input, host_input, num_float * sizeof(float), hipMemcpyHostToDevice);

    float * device_output;
    hipMalloc((void**)&device_output, (block) * sizeof(float));

    // gpu计算reduce

    dim3 grids(block, 1), blocks(thread_per_block, 1);
    reduce_0<<< grids, blocks >>>(device_input, device_output);

    float * gpu_output = (float*)malloc(block * sizeof(float));
    hipMemcpy(gpu_output, device_output, block * sizeof(float), hipMemcpyDeviceToHost);
    
    // cpu计算数组结果
    float* host_output = (float*)malloc(block * sizeof(float));

    for(int i = 0; i < block; i++){
        host_output[i] = 0; 
        for (int j = 0; j < 2 * thread_per_block; j++){
            host_output[i] += host_input[i * 2 * thread_per_block + j];
        }
    }
 
    //cpu 加法
    if (check(gpu_output, host_output, block)){
        printf("answer is right \n");
    }else{
        printf("answer is wrong \n");
    }


    printf("down \n");
    return 0;
}
