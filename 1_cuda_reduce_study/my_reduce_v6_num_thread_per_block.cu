#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <cstdio>

#define thread_per_block 256

template<unsigned int NUM_PRO_BLOCK>
__global__ void reduce_0(float * device_input, float * device_output){

    __shared__ float data[thread_per_block];
    float * block_pointer = device_input + blockIdx.x * NUM_PRO_BLOCK;
    
    int thread_id = threadIdx.x;
    
    data[thread_id] = 0;
    for (int i = 0; i < NUM_PRO_BLOCK / thread_per_block; i++){
        data[thread_id] += block_pointer[thread_id + i * thread_per_block];
    }
    
    __syncthreads();

// #pragma unroll
//     for (int i = blockDim.x / 2; i > 32; i/=2)
//     {
        
//         if(thread_id < i){
            
//             //int index = 2 * thread_id * i;
//             data[thread_id] += data[thread_id + i];

//         }
//         __syncthreads();
//     }

    if(thread_id < 128)
    {
        data[thread_id] += data[thread_id + 128];
        __syncthreads();
    }
    
    if(thread_id < 64)
    {
        data[thread_id] += data[thread_id + 64];
        __syncthreads();
    }    

    if (thread_id < 32){
        data[thread_id] += data[thread_id + 32];
        data[thread_id] += data[thread_id + 16];
        data[thread_id] += data[thread_id + 8];
        data[thread_id] += data[thread_id + 4];
        data[thread_id] += data[thread_id + 2];
        data[thread_id] += data[thread_id + 1];
    }

    if (thread_id == 0)
        device_output[blockIdx.x] = data[0];

    /*
    减少blockIdx.x的一半

    */

}



bool check(float *gpu_result, float * cpu_result, size_t N){

    for(int i = 0; i < N; i++){

        if(abs(gpu_result[i] - cpu_result[i]) > 0.005)
            return false;

    }
    return true;
}

int main(){

    // 生成随机数
    const int num_float = 48 * 1024 * 1024;

    float * host_input = (float*)malloc(num_float * sizeof(float));

    for(int i = 0; i < num_float; i++)
    {
        host_input[i] = 2.0 * (float)drand48() -1.0;
        //printf("host_input[%d]:%f \n", i, host_input[i]);
    }

    //固定block的数量 让一个block中的线程做更多的work
    const int block = 1024;
    const int num_pro_block = num_float / block;


    // 分配device空间
    float * device_input;
    hipMalloc((void**)&device_input, num_float*sizeof(float));
    hipMemcpy(device_input, host_input, num_float * sizeof(float), hipMemcpyHostToDevice);

    float * device_output;
    hipMalloc((void**)&device_output, (block) * sizeof(float));

    // gpu计算reduce

    dim3 grids(block, 1), blocks(thread_per_block, 1);

    reduce_0<num_pro_block><<< grids, blocks >>>(device_input, device_output);

    float * gpu_output = (float*)malloc(block * sizeof(float));
    hipMemcpy(gpu_output, device_output, block * sizeof(float), hipMemcpyDeviceToHost);
    
    // cpu计算数组结果
    float* host_output = (float*)malloc(block * sizeof(float));

    for(int i = 0; i < block; i++){
        host_output[i] = 0; 
        for (int j = 0; j < num_pro_block; j++){
            host_output[i] += host_input[i * num_pro_block + j];
        }
    }
 
    //cpu 加法
    if (check(gpu_output, host_output, block)){
        printf("answer is right \n");
    }else{
        printf("answer is wrong \n");
    }


    printf("down \n");
    return 0;
}
